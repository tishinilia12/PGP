
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>


#define CSC(call) do {				\
	hipError_t res = call;			\
	if (res != hipSuccess) {		\
		fprintf(stderr, "CUDA error %s:%d message: %s\n", __FILE__, __LINE__,	\
				hipGetErrorString(res));	\
		exit(0);							\
	}										\
} while(0)


//__host__ __device__ int add(int a, int b){
//	return a + b;
//}

__global__ void kernel(int *arr, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = gridDim.x * blockDim.x;
	while(idx < n) {
//		assert(idx < 100);
		arr[idx] *= 2;
		idx += offset;
	}
}

int main() {
	int i, n = 100000000;
	int *arr = (int *)malloc(sizeof(int) * n);
	for(i = 0; i < n; i++)
		arr[i] = i;

	int *dev_arr;
	CSC(hipMalloc(&dev_arr, sizeof(int) * n));
	CSC(hipMemcpy(dev_arr, arr, sizeof(int) * n, hipMemcpyHostToDevice));

	float time;
	hipEvent_t start, stop;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));
	CSC(hipEventRecord(start, 0));

	kernel<<<256, 256>>>(dev_arr, n);
	CSC(hipGetLastError());

	CSC(hipEventRecord(stop, 0));
	CSC(hipEventSynchronize(stop));
	CSC(hipEventElapsedTime(&time, start, stop));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));
	printf("time = %f\n", time);

	CSC(hipMemcpy(arr, dev_arr, sizeof(int) * n, hipMemcpyDeviceToHost));

	for(i = n - 100; i < n; i++)
		printf("%d ", arr[i]);
	printf("\n");

	CSC(hipFree(dev_arr));
	free(arr);
	return 0;
}
