
#include <hip/hip_runtime.h>
#include <stdio.h>


#define CSC(call) do {      \
    hipError_t e = call;   \
    if (e != hipSuccess) { \
        fprintf(stderr, "CUDA Error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(0);            \
    }                       \
} while(0)


__global__ void subKernel(double* a, double* b, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Индекс нити
    int offset = gridDim.x * blockDim.x;              // кол-во блоков * размер блока
    while(idx < n) {
        b[idx] = a[idx] * a[idx];
        idx += offset;
    }
}


void sub(double* a, double* b, int n, int numBytes) {

    double* aDev = NULL;
    double* bDev = NULL;

    // Выделяем память на GPU
    CSC(hipMalloc ( (void**)&aDev, numBytes ));
    CSC(hipMalloc ( (void**)&bDev, numBytes ));

    // Задаем конфигурацию запуска нитей
    dim3 threads = 128;
    dim3 blocks = 128;

    CSC(hipMemcpy ( aDev, a, numBytes, hipMemcpyHostToDevice ));

    subKernel<<<blocks, threads>>> (aDev, bDev, n);

    // Копируем результат в память CPU
    CSC(hipMemcpy ( b, bDev, numBytes, hipMemcpyDeviceToHost ));

    // Освобождаем выделенную память
    CSC(hipFree ( aDev ));
    CSC(hipFree ( bDev ));
}


int main() {
    int n;
    scanf("%d", &n);

    int numBytes = n * sizeof(double);

    double* a = (double*) malloc(numBytes);
    double* b = (double*) malloc(numBytes);

    for (int i = 0; i < n; ++i)
        scanf("%lf", a + i);

    sub(a, b, n, numBytes);

    for (int i = 0; i < n; ++i)
        printf("%.10e ", b[i]);
    printf("\n");

    free(a);
    free(b);

    return 0;
}
